#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>

#include <image/image.h>
#include <image/tga.h>

#include "../../CudaBench.h"

__constant__ __device__ unsigned int full = 0xffffffff;

__global__ void threshold(uint8_t *data, uint8_t threshold, int size, int partSize)
{
	int thread = blockDim.x * blockIdx.x + threadIdx.x;
	uint8_t *ptr = data + thread * partSize;
	uint8_t *end = data + min(size, thread * partSize + partSize);

	for (; ptr < end; ++ptr) {
		if (*ptr < threshold) {
			*ptr = 0;
		} else {
			*ptr = 255;
		}
	}
}

__global__ void thresholdSIMD(unsigned int *data, unsigned int threshold, int size)
{
	unsigned int *ptr = data + blockDim.x * blockIdx.x + threadIdx.x;

	*ptr = __vcmpgeu4(*ptr, threshold);
}

int main(int argc, char *argv[])
{
	if (argc != 4) {
		printf("Invalid number of arguments.\n");
		return 1;
	}

	Image *input_image;
	Image *output_image;
	int error;

	if ((error = TGA_readImage(argv[1], &input_image)) != 0) {
		printf("Error when opening image: %d\n", error);
		return 1;
	}

	uint8_t thresholdValue = atoi(argv[2]);
	unsigned int thresholdValue32 = thresholdValue | (thresholdValue << 8) | (thresholdValue << 16) | (thresholdValue << 24);

	if ((error = Image_copy(input_image, &output_image)) != 0) {
		printf("Error when copying image: %d\n", error);
		Image_delete(input_image);
		return 1;
	}

	CudaBench allBench, sendBench, retrieveBench, kernelBench;
	allBench = CudaBench_new();
	sendBench = CudaBench_new();
	retrieveBench = CudaBench_new();
	kernelBench = CudaBench_new();

	int c, size;
	uint8_t *c_data;
	int partSize = 4;
	int threadsPerBlock = 512;
	int blocks = input_image->width * input_image->height / threadsPerBlock / partSize;

	size = input_image->width * input_image->height * sizeof(uint8_t);

	CudaBench_start(allBench);
	hipMalloc(&c_data, size);

	for (c = 0; c < input_image->channels; ++c) {
		CudaBench_start(sendBench);
		hipMemcpy(c_data, input_image->data[c], size, hipMemcpyHostToDevice);
		CudaBench_end(sendBench);

		CudaBench_start(kernelBench);
		thresholdSIMD<<<blocks, threadsPerBlock>>>((unsigned int *) c_data, thresholdValue32, size);
		CudaBench_end(kernelBench);

		CudaBench_start(retrieveBench);
		hipMemcpy(output_image->data[c], c_data, size, hipMemcpyDeviceToHost);
		CudaBench_end(retrieveBench);
	}

	hipFree(c_data);
	CudaBench_end(allBench);

	hipEventSynchronize(allBench.end);

	float timeAll, timeSend, timeKernel, timeRetrieve;

	timeAll = CudaBench_elapsedTime(allBench);
	timeSend = CudaBench_elapsedTime(sendBench);
	timeRetrieve = CudaBench_elapsedTime(retrieveBench);
	timeKernel = CudaBench_elapsedTime(kernelBench);

	printf("All: %f ms\nSend: %f ms\nRetrieve: %f ms\nKernel: %f ms\n", timeAll, timeSend, timeRetrieve, timeKernel);

	CubaBench_delete(allBench);
	CubaBench_delete(sendBench);
	CubaBench_delete(retrieveBench);
	CubaBench_delete(kernelBench);

	if ((error = TGA_writeImage(argv[3], output_image)) != 0) {
		printf("Error when writing image: %d\n", error);
	}

	Image_delete(input_image);
	Image_delete(output_image);

	hipDeviceReset();

	return 0;
}