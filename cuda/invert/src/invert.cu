#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>

#include <image/image.h>
#include <image/tga.h>

__global__ void invert(uint8_t *data, int size, int partSize)
{
	int thread = blockDim.x * blockIdx.x + threadIdx.x;
	int start = thread * partSize;
	int end = min(start + partSize, size);
	int i;

	for (i = start; i < end; ++i) {
		data[i] = 255 - data[i];
	}
}

__constant__ __device__ unsigned int full = 0xffffffff;

__global__ void invertSIMD(unsigned int *data, int size)
{
	int thread = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int *ptr = data + thread;

	*ptr = __vsubss4(full, *ptr);
}

int main(int argc, char *argv[])
{
	if (argc != 3) {
		printf("Invalid number of arguments.\n");
		return 1;
	}

	Image *input_image;
	Image *output_image;
	int error;

	if ((error = TGA_readImage(argv[1], &input_image)) != 0) {
		printf("Error when opening image: %d\n", error);
		return 1;
	}

	if ((error = Image_copy(input_image, &output_image)) != 0) {
		printf("Error when copying image: %d\n", error);
		Image_delete(input_image);
		return 1;
	}

	int partSizes[] = {4, 16, 64, 256, 1024};
	int threadsPerBlocks[] = {32, 96, 192, 384};

	int partSizeIndex = 0;
	int threadsPerBlockIndex = 0;

	for (partSizeIndex = 0; partSizeIndex < 5; ++partSizeIndex) {
		for (threadsPerBlockIndex = 0; threadsPerBlockIndex < 4; ++threadsPerBlockIndex) {
			int c, size;
			uint8_t *c_data;
			int partSize = partSizes[partSizeIndex];
			int threadsPerBlock = threadsPerBlocks[threadsPerBlockIndex];
			int blocks = input_image->width * input_image->height / threadsPerBlock / partSize;

			printf("%d %d\n", partSize, threadsPerBlock);

			size = input_image->width * input_image->height * sizeof(uint8_t);

			hipEvent_t startAll, startChannel, startKernel;
			hipEvent_t stopAll, stopChannel, stopKernel;

			hipEventCreate(&startAll);
			hipEventCreate(&startChannel);
			hipEventCreate(&startKernel);
			hipEventCreate(&stopAll);
			hipEventCreate(&stopChannel);
			hipEventCreate(&stopKernel);

			hipEventRecord(startAll);
			hipMalloc(&c_data, size);

			for (c = 0; c < input_image->channels; ++c) {
				hipEventRecord(startChannel);
				hipMemcpy(c_data, input_image->data[c], size, hipMemcpyHostToDevice);
				hipEventRecord(startKernel);
				invert<<<blocks, threadsPerBlock>>>(c_data, input_image->width * input_image->height, partSize);
				hipEventRecord(stopKernel);
				hipMemcpy(output_image->data[c], c_data, size, hipMemcpyDeviceToHost);
				hipEventRecord(stopChannel);
			}

			hipFree(c_data);
			hipEventRecord(stopAll);

			hipEventSynchronize(stopAll);

			float timeAll, timeChannel, timeKernel;

			hipEventElapsedTime(&timeAll, startAll, stopAll);
			hipEventElapsedTime(&timeChannel, startChannel, stopChannel);
			hipEventElapsedTime(&timeKernel, startKernel, stopKernel);

			hipEventDestroy(startAll);
			hipEventDestroy(startChannel);
			hipEventDestroy(startKernel);
			hipEventDestroy(stopAll);
			hipEventDestroy(stopChannel);
			hipEventDestroy(stopKernel);

			//printf("All: %fms\nChannel: %fms\nKernel: %fms\n", timeAll, timeChannel, timeKernel);
			printf("%f\n\n", timeKernel);
		}
	}

	for (threadsPerBlockIndex = 0; threadsPerBlockIndex < 4; ++threadsPerBlockIndex) {
		int c, size;
		uint8_t *c_data;
		int partSize = 4;
		int threadsPerBlock = threadsPerBlocks[threadsPerBlockIndex];
		int blocks = input_image->width * input_image->height / threadsPerBlock / partSize;

		printf("%d %d\n", partSize, threadsPerBlock);

		size = input_image->width * input_image->height * sizeof(uint8_t);

		hipEvent_t startAll, startChannel, startKernel;
		hipEvent_t stopAll, stopChannel, stopKernel;

		hipEventCreate(&startAll);
		hipEventCreate(&startChannel);
		hipEventCreate(&startKernel);
		hipEventCreate(&stopAll);
		hipEventCreate(&stopChannel);
		hipEventCreate(&stopKernel);

		hipEventRecord(startAll);
		hipMalloc(&c_data, size);

		for (c = 0; c < input_image->channels; ++c) {
			hipEventRecord(startChannel);
			hipMemcpy(c_data, input_image->data[c], size, hipMemcpyHostToDevice);
			hipEventRecord(startKernel);
			invertSIMD<<<blocks, threadsPerBlock>>>((unsigned int *) c_data, input_image->width * input_image->height);
			hipEventRecord(stopKernel);
			hipMemcpy(output_image->data[c], c_data, size, hipMemcpyDeviceToHost);
			hipEventRecord(stopChannel);
		}

		hipFree(c_data);
		hipEventRecord(stopAll);

		hipEventSynchronize(stopAll);

		float timeAll, timeChannel, timeKernel;

		hipEventElapsedTime(&timeAll, startAll, stopAll);
		hipEventElapsedTime(&timeChannel, startChannel, stopChannel);
		hipEventElapsedTime(&timeKernel, startKernel, stopKernel);

		hipEventDestroy(startAll);
		hipEventDestroy(startChannel);
		hipEventDestroy(startKernel);
		hipEventDestroy(stopAll);
		hipEventDestroy(stopChannel);
		hipEventDestroy(stopKernel);

		printf("All: %fms\nChannel: %fms\nKernel: %fms\n", timeAll, timeChannel, timeKernel);
		printf("%f\n\n", timeKernel);
	}

	if ((error = TGA_writeImage(argv[2], output_image)) != 0) {
		printf("Error when writing image: %d\n", error);
	}

	Image_delete(input_image);
	Image_delete(output_image);

	return 0;
}