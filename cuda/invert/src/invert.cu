#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>

#include <image/image.h>
#include <image/tga.h>

#include "../../CudaBench.h"

#define PARTSIZE 4

__global__ void invert(uint8_t *data, int size)
{
	int thread = blockDim.x * blockIdx.x + threadIdx.x;
	int start = thread * PARTSIZE;
	int end = min(start + PARTSIZE, size);
	int i;

	for (i = start; i < end; ++i) {
		data[i] = 255 - data[i];
	}
}

__constant__ __device__ unsigned int full = 0xffffffff;

__global__ void invertSIMD(unsigned int *data, int size)
{
	unsigned int *ptr = data + blockDim.x * blockIdx.x + threadIdx.x;

	*ptr = __vsubss4(full, *ptr);
}

int main(int argc, char *argv[])
{
	if (argc != 3) {
		printf("Invalid number of arguments.\n");
		return 1;
	}

	Image *input_image;
	Image *output_image;
	int error;

	if ((error = TGA_readImage(argv[1], &input_image)) != 0) {
		printf("Error when opening image: %d\n", error);
		return 1;
	}

	if ((error = Image_copy(input_image, &output_image)) != 0) {
		printf("Error when copying image: %d\n", error);
		Image_delete(input_image);
		return 1;
	}

	CudaBench allBench, sendBench, retrieveBench, kernelBench;
	allBench = CudaBench_new();
	sendBench = CudaBench_new();
	retrieveBench = CudaBench_new();
	kernelBench = CudaBench_new();

	int c, size, sizeDevice;
	uint8_t *c_data;
	
	int threadsPerBlock = 128;
	dim3 blocks(input_image->width / 32, input_image->height / 16, 1);

	size = input_image->width * input_image->height * sizeof(uint8_t);
	sizeDevice = size + 4 - (size % 4);

	CudaBench_start(allBench);
	hipMalloc(&c_data, sizeDevice);

	for (c = 0; c < input_image->channels; ++c) {
		CudaBench_start(sendBench);
		hipMemcpy(c_data, input_image->data[c], size, hipMemcpyHostToDevice);
		CudaBench_end(sendBench);

		CudaBench_start(kernelBench);
		invertSIMD<<<blocks, threadsPerBlock>>>((unsigned int *) c_data, input_image->width * input_image->height);
		CudaBench_end(kernelBench);

		CudaBench_start(retrieveBench);
		hipMemcpy(output_image->data[c], c_data, size, hipMemcpyDeviceToHost);
		CudaBench_end(retrieveBench);
	}

	hipFree(c_data);
	CudaBench_end(allBench);

	hipEventSynchronize(allBench.end);

	float timeAll, timeSend, timeKernel, timeRetrieve;

	timeAll = CudaBench_elapsedTime(allBench);
	timeSend = CudaBench_elapsedTime(sendBench);
	timeRetrieve = CudaBench_elapsedTime(retrieveBench);
	timeKernel = CudaBench_elapsedTime(kernelBench);

	printf("All: %f ms\nSend: %f ms\nRetrieve: %f ms\nKernel: %f ms\n", timeAll, timeSend, timeRetrieve, timeKernel);

	CubaBench_delete(allBench);
	CubaBench_delete(sendBench);
	CubaBench_delete(retrieveBench);
	CubaBench_delete(kernelBench);

	if ((error = TGA_writeImage(argv[2], output_image)) != 0) {
		printf("Error when writing image: %d\n", error);
	}

	Image_delete(input_image);
	Image_delete(output_image);

	hipDeviceReset();

	return 0;
}